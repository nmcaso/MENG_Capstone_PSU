#include "hip/hip_runtime.h"
#include "C:/Program Files/MATLAB/R2022b/extern/include/mex.h"
#include "C:/Program Files/MATLAB/R2022b/toolbox/parallel/gpu/extern/include/gpu/mxGPUArray.h"
#include <chrono>

#define timer       std::chrono::high_resolution_clock
#define timertime   std::chrono::high_resolution_clock::time_point
#define timecast    std::chrono::duration_cast<std::chrono::microseconds>
#define timesecs    std::chrono::microseconds

//a convenient structure
typedef struct sizes {
            size_t i;
            size_t j;
            size_t k;
            size_t imax;
            size_t jmax;
            size_t kmax;
            size_t xysize;
            size_t M_numel;
} sizes;

//a device function to reduce the last several warps of the reduction faster.
__device__ void warpreduce(volatile double* s_matrix, int thread_vector) {
    s_matrix[thread_vector] += s_matrix[thread_vector + 32];
    s_matrix[thread_vector] += s_matrix[thread_vector + 16];
    s_matrix[thread_vector] += s_matrix[thread_vector + 8];
    s_matrix[thread_vector] += s_matrix[thread_vector + 4];    
    s_matrix[thread_vector] += s_matrix[thread_vector + 2];
    s_matrix[thread_vector] += s_matrix[thread_vector + 1];
}

//kernel Creates a beamformed image array on the GPU.
__global__ void DAS_Index_GPU(const unsigned int* indmat, const double* rfdata, double* img3d, double* img, sizes sz) {    
    // define device variables
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    //memory index
    if(z < sz.kmax && y < sz.jmax && x < sz.imax) {
        int M_index             = x + y * sz.imax + z * sz.xysize;
        int Img_Index           = z + x * sz.kmax + y * sz.kmax*sz.imax;
        *(img3d + Img_Index) = *(rfdata + *(indmat + M_index));
    }
}

__global__ void DAS_3DSUM(double* matrix_3d, double* matrix_2d, sizes sz) {

    extern __shared__ double shared_matrix_data[];

    int thread_vector   = threadIdx.x; //1:512
    int all_threads     = blockIdx.x * blockDim.x*2 + threadIdx.x; //1:512 + 1:160k*512

    //first add during global load, we've done the first add from 512 - 256 elements
    shared_matrix_data[thread_vector] = matrix_3d[all_threads] + matrix_3d[all_threads + blockDim.x];
    __syncthreads();
    
    //interleaved addition for 2 interations (no loop = no overhead)
    if(thread_vector < 128) {
        shared_matrix_data[thread_vector] += shared_matrix_data[thread_vector + 128];} __syncthreads();
    if(thread_vector < 64) {
        shared_matrix_data[thread_vector] += shared_matrix_data[thread_vector + 64];} __syncthreads();

    //run the unwrapped warp reduction function
    if(thread_vector < 32) warpreduce(shared_matrix_data, thread_vector);

    //copy to global memory
    if(thread_vector == 0) matrix_2d[blockIdx.x] = shared_matrix_data[0];
}

//wrapper into MATLAB
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {

    mxInitGPU();
    const char*         const errId = "parallel:gpu:gpudasindex:InvalidInput"; //error message
    
    //Input Error handling
    if (nrhs!=5) {
        mexErrMsgIdAndTxt(errId, "Expected 5 inputs: M, rfdata, thread/block x, thread/block y, thread/block z");
    } 
    if (nlhs!=1 && nlhs!=0) {
        mexErrMsgIdAndTxt(errId, "Invalid number of output arguments, only 1 allowed.");
    }
    if (!mxIsGPUArray(prhs[0])) {
        mexErrMsgIdAndTxt(errId, "Index Matrix M must be a GPU array.");
    } 
    if (!mxIsGPUArray(prhs[1])) {
        mexErrMsgIdAndTxt(errId, "rfdata must be a GPU array.");
    } 
    if (!mxIsDouble(prhs[2]) || !mxIsDouble(prhs[3]) || !mxIsDouble(prhs[4])) {
        mexErrMsgIdAndTxt(errId, "Block dimensions must be of datatype 'double'");
    }

    // Load mx gpu array objects
    const mxGPUArray*   M           = mxGPUCreateFromMxArray(prhs[0]);
    const mxGPUArray*   rfdata      = mxGPUCreateFromMxArray(prhs[1]);
    const mwSize*       dims_3      = mxGPUGetDimensions(M);
    const mwSize        xysz[2]     = {dims_3[0], dims_3[1]};

    sizes sz1 = {
    //define block and thread sizes
    (unsigned long long)*(double*)mxGetData(prhs[2]),
    (unsigned long long)*(double*)mxGetData(prhs[3]),
    (unsigned long long)*(double*)mxGetData(prhs[4]),
    
    //Get the number of x/y/z threads
    dims_3[1],
    dims_3[0],
    dims_3[2],
    dims_3[0] * dims_3[1],
    dims_3[2] * dims_3[0] * dims_3[1]
    };

    //round up to the nearest integer of the dimension length divided by the thread   dimension
    int blockx = sz1.imax/sz1.i + (sz1.imax % sz1.i != 0);
    int blocky = sz1.jmax/sz1.j + (sz1.jmax % sz1.j != 0);
    int blockz = sz1.kmax/sz1.k + (sz1.kmax % sz1.k != 0);

    //grid and block 3D arrays
    dim3    threads(sz1.i, sz1.j, sz1.k);
    dim3    block(blockx, blocky, blockz);
    size_t  total_threads = sz1.i * sz1.j * sz1.k;

    const mwSize* mnum = &sz1.M_numel;
    //Verify that inputs are correct classes before extracting the pointer.
    if (mxGPUGetClassID(M) != mxUINT32_CLASS) {
        mexErrMsgIdAndTxt(errId, "Index Matrix M must have class 'uint32'");
    }
    if (mxGPUGetClassID(rfdata) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, "rfdata must have class 'double'");
    } 
    if (total_threads > 1024 || total_threads <= 1) {
        mexErrMsgIdAndTxt(errId, "Block dimensions product must be between 1 and 1024");
    }

    // Extract a pointer to the input data on the device.
    const unsigned int*     M_dvc       = (const unsigned int*) (mxGPUGetDataReadOnly(M));
    const double*           rfdata_dvc  = (const double*)       (mxGPUGetDataReadOnly(rfdata));

    // Create a GPUArray to hold the result and get its underlying pointer.
    mxGPUArray* img_3d      = mxGPUCreateGPUArray(1, mnum, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double*     img3_dvc    = (double*)(mxGPUGetData(img_3d));

    mxGPUArray* img_2d      = mxGPUCreateGPUArray(2, xysz, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double*     img2_dvc    = (double*)(mxGPUGetData(img_2d));

    timertime start8 = timer::now();
    // Call the kernel
    DAS_Index_GPU<<<block, threads>>>(M_dvc, rfdata_dvc, img3_dvc, img2_dvc, sz1);
    hipDeviceSynchronize();

    //Call the other kernel
    DAS_3DSUM <<<sz1.xysize, 256, 256*sizeof(double)>>>(img3_dvc, img2_dvc, sz1);
    hipDeviceSynchronize();

    timertime stop8 = timer::now();
    timesecs indexgpu = timecast(stop8 - start8);
    printf("Kernel Index: %d\n",indexgpu);
    
    //Get the result as a gpuArray
    plhs[0] = mxGPUCreateMxArrayOnGPU(img_2d); //takes about 38 microsecs

    //cleanup
    mxGPUDestroyGPUArray(M);
    mxGPUDestroyGPUArray(rfdata);
    mxGPUDestroyGPUArray(img_3d);
    mxGPUDestroyGPUArray(img_2d);
    //cleanup takes about 8 microsecs
}
