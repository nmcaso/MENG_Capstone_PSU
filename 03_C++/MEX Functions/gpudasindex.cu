#include "hip/hip_runtime.h"
#include "C:/Program Files/MATLAB/R2022b/extern/include/mex.h"
#include "C:/Program Files/MATLAB/R2022b/toolbox/parallel/gpu/extern/include/gpu/mxGPUArray.h"
#include <chrono>

#define timer       std::chrono::high_resolution_clock
#define timertime   std::chrono::high_resolution_clock::time_point
#define timecast    std::chrono::duration_cast<std::chrono::microseconds>
#define timesecs    std::chrono::microseconds

//a convenient structure
typedef struct sizes {
            size_t imax;
            size_t jmax;
            size_t kmax;
            size_t xysize;
            size_t M_numel;
} sizes;

//a device function to reduce the last several warps of the reduction faster.
__device__ void warpreduce(volatile double* s_matrix, int thread_vector) {
    s_matrix[thread_vector] += s_matrix[thread_vector + 32];
    s_matrix[thread_vector] += s_matrix[thread_vector + 16];
    s_matrix[thread_vector] += s_matrix[thread_vector + 8];
    s_matrix[thread_vector] += s_matrix[thread_vector + 4];    
    s_matrix[thread_vector] += s_matrix[thread_vector + 2];
    s_matrix[thread_vector] += s_matrix[thread_vector + 1];
}

//kernel Creates a beamformed image array on the GPU.
__global__ void DAS_Index_GPU(const unsigned int* indmat, const double* rfdata, double* img3d) {    

    //nice easy 1D kernel
    int z = threadIdx.x;
    int xy = blockIdx.x;
    int xyz = xy * blockDim.x + z; 

    img3d[xyz] = rfdata[indmat[xyz]];
}

//going to have to transpose this to be faster. I'll do that tomorrow. take a look at https://github.com/shwina/cuper/blob/master/cuTranspose/transpose3d.cu, dev_transpose_102_in_place

//kernel to take the sum in the coalesced first dimension of the 3D array
__global__ void DAS_3DSUM(double* matrix_3d, double* matrix_2d) {

    extern __shared__ double shared_matrix_data[];

    int thread_vector   = threadIdx.x; //1:256
    int all_threads     = blockIdx.x * blockDim.x*2 + threadIdx.x; //1:256 + 1:160k*256*2

    //first add during global load
    shared_matrix_data[thread_vector] = matrix_3d[all_threads] + matrix_3d[all_threads + blockDim.x];
    __syncthreads();
    
    //interleaved addition for 2 interations (no loop = no overhead)
    if(thread_vector < 128) {
        shared_matrix_data[thread_vector] += shared_matrix_data[thread_vector + 128];} __syncthreads();
    if(thread_vector < 64) {
        shared_matrix_data[thread_vector] += shared_matrix_data[thread_vector + 64];} __syncthreads();

    //run the unwrapped warp reduction function
    if(thread_vector < 32) warpreduce(shared_matrix_data, thread_vector);

    //copy to global memory
    if(thread_vector == 0) matrix_2d[blockIdx.x] = shared_matrix_data[0];
}

//wrapper into MATLAB
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {

    mxInitGPU();
    const char*         const errId = "parallel:gpu:gpudasindex:InvalidInput"; //error message

    //Input Error handling
    if (nrhs!=2) {
        mexErrMsgIdAndTxt(errId, "Expected 2 inputs: Index Matrix M (3D), rfdata matrix (2D)");
    } 
    if (nlhs!=1 && nlhs!=0) {
        mexErrMsgIdAndTxt(errId, "Invalid number of output arguments, only 1 allowed.");
    }
    if (!mxIsGPUArray(prhs[0])) {
        mexErrMsgIdAndTxt(errId,"Index Matrix M must be a gpuArray.");
    }
    if (!mxIsGPUArray(prhs[1])) {
        mexErrMsgIdAndTxt(errId,"rfdata must be a gpuArray.");
    }

    // Load mx gpu array objects
    const mxGPUArray*   M           = mxGPUCreateFromMxArray(prhs[0]);
    const mxGPUArray*   rfdata      = mxGPUCreateFromMxArray(prhs[1]);
    const mwSize*       dims_3      = mxGPUGetDimensions(M);
    const mwSize        rfdims      = mxGPUGetNumberOfDimensions(rfdata);
    const mwSize        mdims       = mxGPUGetNumberOfDimensions(M);

    if (mdims != 3) {
        mexErrMsgIdAndTxt(errId, "Index Matrix M must have 3 dimensions");
    }

    const mwSize        xysz[2]     = {dims_3[0], dims_3[1]};

    sizes sz1 = {
    //define block and thread sizes
   
    //Get the number of x/y/z threads
    dims_3[1],
    dims_3[0],
    dims_3[2],
    dims_3[0] * dims_3[1],
    dims_3[2] * dims_3[0] * dims_3[1]
    };

    const mwSize* mnum = &sz1.M_numel;

    //Verify that inputs are correct classes before extracting the pointer.
    if (rfdims != 2) {
        mexErrMsgIdAndTxt(errId, "rfdata input must have 2 dimensions (more than 1 frame per function call is not supported)");
    }
    if (mxGPUGetClassID(M) != mxUINT32_CLASS) {
        mexErrMsgIdAndTxt(errId, "Index Matrix M must have class 'uint32'");
    }
    if (mxGPUGetClassID(rfdata) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, "rfdata must have class 'double'");
    } 
    if(dims_3[2] != 512) {
        mexErrMsgIdAndTxt(errId, "Incorrect number of sensors detected. The third dimension of the index matrix should be the number of sensors, 512. Contact the developer at caso.nathan@gmail.com if you are trying to use a different number of sensors.");
    }

    // Extract a pointer to the input data on the device.
    const unsigned int*     M_dvc       = (const unsigned int*) (mxGPUGetDataReadOnly(M));
    const double*           rfdata_dvc  = (const double*)       (mxGPUGetDataReadOnly(rfdata));

    // GPU Arrays and device-side pointers for the 3-D image and 2-D image
    mxGPUArray* img_3d      = mxGPUCreateGPUArray(1, mnum, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double*     img3_dvc    = (double*)(mxGPUGetData(img_3d));

    mxGPUArray* img_2d      = mxGPUCreateGPUArray(2, xysz, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    double*     img2_dvc    = (double*)(mxGPUGetData(img_2d));

    //Call the DAS Indexing kernel
    // DAS_Index_GPU<<<block, threads>>>(M_dvc, rfdata_dvc, img3_dvc, img2_dvc, sz1);
    DAS_Index_GPU <<<sz1.xysize, 512>>> (M_dvc, rfdata_dvc, img3_dvc);
    hipDeviceSynchronize();

    //Call the flattening kernel
    DAS_3DSUM <<<sz1.xysize, 256, 256*sizeof(double)>>>(img3_dvc, img2_dvc);
    hipDeviceSynchronize();
    
    //Get the result as a gpuArray
    plhs[0] = mxGPUCreateMxArrayOnGPU(img_2d); //takes about 38 microsecs

    //cleanup (takes about 8 microseconds)
    mxGPUDestroyGPUArray(M);
    mxGPUDestroyGPUArray(rfdata);
    mxGPUDestroyGPUArray(img_3d);
    mxGPUDestroyGPUArray(img_2d);
}
