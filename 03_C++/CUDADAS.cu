#include "hip/hip_runtime.h"
#include "Header Files\CUDADAS.cuh"

using namespace std;

 struct sizes {
                        int i;
                        int j;
                        int k;
                        int imax;
                        int jmax;
                        int kmax;
                        int xysize;
    };

//Creates a beamformed image array on the GPU.
__global__ void DAS_Index_GPU(float* rfptr, unsigned int* indmat, double* reconptr, sizes sz) {    
    
    //preallocate
    int x               = threadIdx.x + blockDim.x*blockIdx.x;
    int y               = threadIdx.y + blockDim.y*blockIdx.y;
    int z               = threadIdx.z + blockDim.z*blockIdx.z;
    unsigned int        final_index;

    //memory indexing (works correctly)
     if(x < sz.imax && y < sz.jmax && z < sz.kmax) {
        final_index = *(indmat + x + y*sz.imax + z*sz.xysize);
        *(reconptr+x+sz.imax*y + z*sz.xysize) = *(rfptr+final_index);
     }
}

//This function takes the GPU array form DAS_Index_GPU (in 3D) and does an average along the z-dimension, thereby flattening it to 2 dimensions for image viewing
__global__ void DAS_Index_Flatten(double* recon_3d, double* recon_2d, sizes sz) {

    //preallocate
    int x               = threadIdx.x + blockDim.x*blockIdx.x;
    int y               = threadIdx.y + blockDim.y*blockIdx.y;
    int z               = threadIdx.z + blockDim.z*blockIdx.z;

    // does not coalesce properly, see gpudasindex.cu mex function for the correct function)
    if(x < sz.imax && y < sz.jmax && z < sz.kmax) {
        *(recon_2d+x+sz.imax*y + z*sz.xysize) += *(recon_3d + x + sz.imax*y + z*sz.xysize);
    }

}

//Wrapper function to call from C++ Code that will initialize variables on the GPU and call the kernel with proper syntax.
void indexgpuwrapper(Dataset data2, IndexMatrix indmat, double* cpureconptr) {
    
    struct              sizes sz1;

    //define block and thread sizes
    sz1.imax            = indmat.M_cols;
    sz1.jmax            = indmat.M_rows;
    sz1.kmax            = indmat.M_depth;
    sz1.xysize          = sz1.imax*sz1.jmax;
    
    //Get the number of threads in each direction
    sz1.i               = 16;
    sz1.j               = 16;
    sz1.k               = 4;
        
    //round up to the nearest integer of the dimension length divided by the thread dimension
    int blockx          = indmat.M_rows/sz1.i + (indmat.M_rows % sz1.i != 0);
    int blocky          = indmat.M_cols/sz1.j + (indmat.M_cols % sz1.j != 0);
    int blockz          = indmat.M_depth/sz1.k +(indmat.M_depth % sz1.k != 0);

    //grid and block 3D arrays
    dim3                grid(sz1.i, sz1.j, sz1.k);
    dim3                block(blockx, blocky, blockz);

    //preallocate a GPU pointer and make convenient size variables
    int     rfsize      = data2.rfdata_cols*data2.rfdata_rows;
    double*  recontemp  = new double [indmat.M_numel]();
    double*  reconptr   = new double [sz1.xysize]();
    unsigned int* gpu_index   = new unsigned int [indmat.M_numel];
    float*  gpu_rf      = new float [rfsize];

    // preallocate space on the GPU for the big variables
    hipMallocManaged   (&reconptr,     indmat.M_numel  *sizeof(double)); //**use if outputting index matrix
    hipMallocManaged   (&recontemp,    indmat.M_numel  *sizeof(double));
    hipMallocManaged   (&gpu_rf,       rfsize          *sizeof(float));
    hipMallocManaged   (&gpu_index,    indmat.M_numel  *sizeof(int));

    //load the big variables onto the GPU
    hipMemcpy          (gpu_index,     indmat.M,       indmat.M_numel*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy          (gpu_rf,        data2.rfptr,    rfsize*sizeof(float),        hipMemcpyHostToDevice);

    //set up a number of runs to perform
    int                 numavgs = 1;

    auto start3         = chrono::high_resolution_clock::now();

    //run the DAS algorithm on the GPU a number of times equal to numavgs.
    for(int i = 0; i < numavgs; ++i) {
        DAS_Index_GPU       <<< block, grid >>>     (gpu_rf, gpu_index, reconptr, sz1);
        // DAS_Index_Flatten   <<<block,grid>>>        (recontemp,reconptr,sz1); //comment this line if you're outputting the index matrix
        hipDeviceSynchronize                       ();
    }

    auto stop3          = chrono::high_resolution_clock::now();
    auto duration3      = std::chrono::duration_cast<std::chrono::microseconds>(stop3 - start3);
    cout << "GPU Computing Time: " << static_cast<float>(duration3.count())/static_cast<float>(numavgs) << " ms" << endl;

    auto start4         = chrono::high_resolution_clock::now();
    // Copy the GPU array back to the CPU
    hipMemcpy          (cpureconptr, reconptr, indmat.M_numel*sizeof(double), hipMemcpyDeviceToHost); //copies the index matrix out
    hipFree            (reconptr);
    hipFree            (gpu_rf);
    hipFree            (gpu_index);
    hipFree            (recontemp);
    auto stop4          = chrono::high_resolution_clock::now();
    auto duration4      = std::chrono::duration_cast<std::chrono::microseconds>(stop4 - start4);
    cout << "GPU back to CPU and Free Memory time: " << static_cast<float>(duration4.count()) << " ms" << endl;
}